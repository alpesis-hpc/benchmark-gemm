#include "hip/hip_runtime.h"
#include "kernels_cuda.cuh"

/*
 * C := alpha * C
 */
__global__ void VectorMulKernel(int m, int n,
                                float alpha,
                                float * device_a, int lda)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
 
  if (i < m*n)
  {
    device_a[i] = alpha * device_a[i];
  }
}
