#include <stdlib.h>

#include <unity.h>
#include <hip/hip_runtime.h>

#include "gemm_cpu.h"
#include "gemm_cuda.h"


void test_gemm_cuda (void)
{
  const unsigned int m = 20;
  const unsigned int n = 20;
  const unsigned int k = 20;
  const float alpha = 0.0f;
  const float beta = 2.0f;
  const unsigned int lda = k;
  const unsigned int ldb = n;
  const unsigned int ldc = n;
  float * host_a = (float*)malloc(m*k*sizeof(float));
  float * host_b = (float*)malloc(k*n*sizeof(float));
  float * host_c_cpu = (float*)malloc(m*n*sizeof(float));
  float * host_c_gpu = (float*)malloc(m*n*sizeof(float));

  size_t i;
  for (i = 0; i < m*k; ++i) { host_a[i] = (float)rand() / (float)RAND_MAX; };
  for (i = 0; i < k*n; ++i) { host_b[i] = (float)rand() / (float)RAND_MAX; };
  for (i = 0; i < m*n; ++i) { host_c_cpu[i] = 1.0f; host_c_gpu[i] = 1.0f; };

  float * device_a;
  float * device_b;
  float * device_c;
  hipMalloc (&device_a, m*k*sizeof(float));
  hipMalloc (&device_b, k*n*sizeof(float));
  hipMalloc (&device_c, m*n*sizeof(float));
  hipMemcpy (device_a, host_a, m*k*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (device_b, host_b, k*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy (device_c, host_c_gpu, m*n*sizeof(float), hipMemcpyHostToDevice);
  gemm_cuda (0, 0, m, n, k, alpha, device_a, lda, device_b, ldb, beta, device_c, ldc);
  hipMemcpy (host_c_gpu, device_c, m*n*sizeof(float), hipMemcpyDeviceToHost);

  gemm_cpu (0, 0, m, n, k, alpha, host_a, lda, host_b, ldb, beta, host_c_cpu, ldc);

  for (int i = 0; i < m*n; ++i)
    TEST_ASSERT_EQUAL_FLOAT (host_c_cpu[i], host_c_gpu[i]);

  hipFree (device_a);
  hipFree (device_b);
  hipFree (device_c);
  free (host_a);
  free (host_b);
  free (host_c_cpu);
  free (host_c_gpu);
}


int main (void)
{
  UNITY_BEGIN ();
  RUN_TEST (test_gemm_cuda);
  return UNITY_END ();
}

